#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#define PSA_TYPE_COLLECT 1
#define PSA_TYPE_DISTRIBUTE 2

const int CUDA_NUM_THREADS = 512;

inline int GET_BLOCKS(const int N) {
   return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename T>
__global__ void psa_collect_forward_kernel(const float *h, float *out, int num, int height, int width) {
    const int out_h = 2 * height - 1;
    const int out_w = 2 * width - 1;
    const int half_out_h = (out_h - 1) / 2;
    const int half_out_w = (out_w - 1) / 2;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = num * height * width;

    for (int i = x; i < nthreads; i += blockDim.x * gridDim.x) {
        const int w = i % width;
        const int h = (i / width) % height;
        const int n = i / width / height;

        // effective mask region : [hstart, hend) x [wstart, wend) with out-indexed
        const int hstart = max(0, half_out_height - h);
        const int hend = min(out_h, height + half_out_h - h);
        const int wstart = max(0, half_out_width - w);
        const int wend = min(out_w, width + half_out_w - w);

        // (hidx, widx) with out-indexed
        // (hidx + h - half_mask_H_, widx + w - half_mask_W_) with feature-indexed
        for (int hidx = hstart; hidx < hend; hidx++) {
            for (int widx = wstart; widx < wend; widx++) {
                out[(n * height * width + (hidx + h - half_out_h) * width + (widx + w - half_out_w)) * height * width + h * width + w] =
                    h[((n * out_h * out_w + hidx * out_w + widx) * height + h) * width + w];
            }
        }
    }
}

template <typename T>
__global__ void psa_distribute_forward_kernel(const float *h, float *out, int num, int height, int width) {
    const int out_h = 2 * height - 1;
    const int out_w = 2 * width - 1;
    const int half_out_h = (out_h - 1) / 2;
    const int half_out_w = (out_w - 1) / 2;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = num * height * width;

    for (int i = x; i < nthreads; i += blockDim.x * gridDim.x) {
        const int w = i % width;
        const int h = (i / width) % height;
        const int n = i / width / height;

        // effective mask region : [hstart, hend) x [wstart, wend) with out-indexed
        const int hstart = max(0, half_out_height - h);
        const int hend = min(out_h, height + half_out_h - h);
        const int wstart = max(0, half_out_width - w);
        const int wend = min(out_w, width + half_out_w - w);

        // (hidx, widx) with out-indexed
        // (hidx + h - half_mask_H_, widx + w - half_mask_W_) with feature-indexed
        for (int hidx = hstart; hidx < hend; hidx++) {
            for (int widx = wstart; widx < wend; widx++) {
                out[(n * height * width + h * width + w) * height * width + (hidx + h - half_out_h) * width + (widx + w - half_out_w)] =
                    h[((n * out_h * out_w + hidx * out_w + widx) * height + h) * width + w];
            }
        }
    }
}

template <typename T>
__global__ void psa_collect_backward_kernel(const float *dout, float *dh, int num, int height, int width) {
    const int out_h = 2 * height - 1;
    const int out_w = 2 * width - 1;
    const int half_out_h = (out_h - 1) / 2;
    const int half_out_w = (out_w - 1) / 2;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = num * height * width;

    for (int i = x; i < nthreads; i += blockDim.x * gridDim.x) {
        const int w = i % width;
        const int h = (i / width) % height;
        const int n = i / width / height;

        // effective mask region : [hstart, hend) x [wstart, wend) with out-indexed
        const int hstart = max(0, half_out_height - h);
        const int hend = min(out_h, height + half_out_h - h);
        const int wstart = max(0, half_out_width - w);
        const int wend = min(out_w, width + half_out_w - w);

        // (hidx, widx) with out-indexed
        // (hidx + h - half_mask_H_, widx + w - half_mask_W_) with feature-indexed
        for (int hidx = hstart; hidx < hend; hidx++) {
            for (int widx = wstart; widx < wend; widx++) {
                dh[((h * out_h * out_w + hidx * out_w + widx) * height + h) * width + w] =
                    dout[(n * height * width + (hidx + h - half_out_h) * width + (widx + w - half_out_w)) * height * width + h * width + w];
            }
        }
    }
}

template <typename T>
__global__ void psa_distribute_backward_kernel(const float *dout, float *dh, int num, int height, int width) {
    const int out_h = 2 * height - 1;
    const int out_w = 2 * width - 1;
    const int half_out_h = (out_h - 1) / 2;
    const int half_out_w = (out_w - 1) / 2;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = num * height * width;

    for (int i = x; i < nthreads; i += blockDim.x * gridDim.x) {
        const int w = i % width;
        const int h = (i / width) % height;
        const int n = i / width / height;

        // effective mask region : [hstart, hend) x [wstart, wend) with out-indexed
        const int hstart = max(0, half_out_height - h);
        const int hend = min(out_h, height + half_out_h - h);
        const int wstart = max(0, half_out_width - w);
        const int wend = min(out_w, width + half_out_w - w);

        // (hidx, widx) with out-indexed
        // (hidx + h - half_mask_H_, widx + w - half_mask_W_) with feature-indexed
        for (int hidx = hstart; hidx < hend; hidx++) {
            for (int widx = wstart; widx < wend; widx++) {
                dh[((n * out_h * out_w + hidx * out_w + widx) * height + h) * width + w] =
                    dout[(n * height * width + h * width + w) * height * width + (hidx + h - half_out_h) * width + (widx + w - half_out_w)];
            }
        }
    }
}

at::Tensor psa_forward_cuda(const at::Tensor& h, const int forward_type) {
    AT_ASSERTM(h.type().is_cuda(), "input must be a CUDA tensor");

    auto n = h.size(0);
    auto c = h.size(1);
    auto h = h.size(2);
    auto w = h.size(3);

    at::Tensor out = at::zeros({n, h * w, h * w}, h.options());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    int nthreads = n * h * w;

    switch (forward_type) {
    case PSA_TYPE_COLLECT:
        AT_DISPATCH_FLOATING_TYPES(h.type(), "psa_forward", [&] {
            psa_collect_forward_kernel<scalar_t><<<GET_BLOCKS(nthreads), CUDA_NUM_THREADS, 0, stream>>>(
                h.contiguous().data<scalar_t>(),
                out.contiguous().data<scalar_t>(),
                n, h, w);
        });
        break;
    case PSA_TYPE_DISTRIBUTE:
        AT_DISPATCH_FLOATING_TYPES(h.type(), "psa_forward", [&] {
            psa_distribute_forward_kernel<scalar_t><<<GET_BLOCKS(nthreads), CUDA_NUM_THREADS, 0, stream>>>(
                h.contiguous().data<scalar_t>(),
                out.contiguous().data<scalar_t>(),
                n, h, w);
        });
        break;
    }
    THCudaCheck(hipGetLastError());
    return out;
}

at::Tensor psa_backward_cuda(const at::Tensor& dout, const at::Tensor& h, const int forward_type) {
    AT_ASSERTM(dout.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(h.type().is_cuda(), "input must be a CUDA tensor");

    auto n = h.size(0);
    auto c = h.size(1);
    auto h = h.size(2);
    auto w = h.size(3);

    at::Tensor dh = at::zeros_like(h);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    int nthreads = n * h * w;

    switch (forward_type) {
    case PSA_TYPE_COLLECT:
        AT_DISPATCH_FLOATING_TYPES(h.type(), "psa_backward", [&] {
            psa_collect_backward_kernel<scalar_t><<<GET_BLOCKS(nthreads), CUDA_NUM_THREADS, 0, stream>>>(
                dout.contiguous().data<scalar_t>(),
                dh.contiguous().data<scalar_t>(),
                n, h, w);
        });
        break;
    case PSA_TYPE_DISTRIBUTE:
        AT_DISPATCH_FLOATING_TYPES(h.type(), "psa_backward", [&] {
            psa_distribute_backward_kernel<scalar_t><<<GET_BLOCKS(nthreads), CUDA_NUM_THREADS, 0, stream>>>(
                dout.contiguous().data<scalar_t>(),
                dh.contiguous().data<scalar_t>(),
                n, h, w);
        });
        break;
    }
    THCudaCheck(hipGetLastError());
    return dh;
}